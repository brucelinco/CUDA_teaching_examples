#include "hip/hip_runtime.h"
#include "book.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#define N   1024*1024

__global__ void add( int *colors, int *bucket ) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        int c = colors[tid];
		atomicAdd(&bucket[c], 1);
		//bucket[c]++;
        tid += blockDim.x * gridDim.x;	
    }
}

int main( void ) {
    int *colors;
	int bucket[256]={0};
	int golden[256]={0};
    int *dev_colors, *dev_bucket;
    int i;
 
 // allocate the memory on the CPU
    colors = (int*)malloc( N * sizeof(int) );
 
    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_colors, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_bucket, 256 * sizeof(int) ) );


    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        colors[i] = rand()%256;
		golden[colors[i]]++;
    }
    
	
    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_colors, colors, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_bucket, bucket, 256 * sizeof(int),
                              hipMemcpyHostToDevice ) );
    // Get start time event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
	
    add<<<65535,256>>>( dev_colors, dev_bucket );
	 
	// Get stop time event    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 
    // Compute execution time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
	
	//check cuda error
    hipError_t status = hipGetLastError();
    if ( hipSuccess != status ){
        fprintf(stderr, "Error: %s\n", hipGetErrorString(status));
        exit(1) ;
    }
	
    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( bucket, dev_bucket, 256 * sizeof(int),
                              hipMemcpyDeviceToHost ) );

    // verify that the GPU did the work we requested
    bool success = true;
    for (int i=0; i<256; i++) {
        if (golden[i] != bucket[i]) {
            printf( "Error at bucket[%d]\n", i);
            success = false;
        }
    }
    if (success)    printf( "We did it!\n" );

    // free the memory we allocated on the GPU
    HANDLE_ERROR( hipFree( dev_colors ) );
    HANDLE_ERROR( hipFree( dev_bucket ) );
 

    // free the memory we allocated on the CPU
    free( colors );


    return 0;
}

