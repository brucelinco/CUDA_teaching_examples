#include "hip/hip_runtime.h"
#include "book.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#define N   32*32*32*32 

__global__ void add( int *a, int *b, int *c ) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int tid = x + y * gridDim.x * blockDim.x;
	while (tid < N){
	   c[tid] = a[tid] + b[tid];
	   tid += gridDim.x * gridDim.y * blockDim.x * blockDim.y;
	}
}

int main( void ) {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;

    a = (int*)malloc(sizeof(int) * N);
    b = (int*)malloc(sizeof(int) * N);
    c = (int*)malloc(sizeof(int) * N);

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

    // fill the arrays 'a' and 'b' on the CPU
	srand ( time(NULL) );
    for (int i=0; i<N; i++) {
        a[i] = rand()%256;
        b[i] = rand()%256;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    // Get start time event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    dim3 grid(16, 16);
    dim3 block(32,32 );

    add<<<grid,block>>>( dev_a, dev_b, dev_c );

	// Get stop time event    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 
    // Compute execution time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
	
    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int),
                              hipMemcpyDeviceToHost ) );

    // display the results
	/*
	for (int i=0; i<N; i++) {
        printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    }
	*/
	bool success = true;
    for (int i=0; i<N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
            success = false;
        }
    }
    if (success)    printf( "We did it!\n" );

    // free the memory allocated on the GPU
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );
    free(a);
    free(b);
    free(c);
    return 0;
}
