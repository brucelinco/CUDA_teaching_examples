#include "hip/hip_runtime.h"
#include "book.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#define N   64*1024*1024

__global__ void add( int *a, int *b, int *c, int seg_size ) {
    int tid = (threadIdx.x + blockIdx.x * blockDim.x) * seg_size;
    while (tid < N) {
	for(int i=tid; i<tid+seg_size; i++){
        	c[i] = a[i] + b[i];
	}
        
        tid += (blockDim.x * gridDim.x)*seg_size;
    }
}

int main( int argc, char* argv[] ) {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;

    if(argc!=2){
	printf("using command %s [segment_size]\n", argv[0]);
 	exit(1);
    }
    // allocate the memory on the CPU
    a = (int*)malloc( N * sizeof(int) );
    b = (int*)malloc( N * sizeof(int) );
    c = (int*)malloc( N * sizeof(int) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = 2 * i;
	c[i] = 0;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    // Get start time event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    int threadsPerBlock = 256;
    int segment_size = atoi(argv[1]);
    int segment_num = (N + segment_size -1)/segment_size;
    int blocksPerGrid = (segment_num + threadsPerBlock -1)/threadsPerBlock;
    if(blocksPerGrid > 65535)
	blocksPerGrid = 65535;
    printf("threadsPerBlock: %d\n", threadsPerBlock);
    printf("segment_size: %d\n", segment_size);
    printf("segment_number: %d\n", segment_num);
    printf("blocksPerGrid: %d\n", blocksPerGrid);


    add<<<blocksPerGrid, threadsPerBlock>>>( dev_a, dev_b, dev_c, segment_size );
	 
	// Get stop time event    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 
    // Compute execution time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
	
	//check cuda error
    hipError_t status = hipGetLastError();
    if ( hipSuccess != status ){
        fprintf(stderr, "Error: %s\n", hipGetErrorString(status));
        exit(1) ;
    }
	
    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int),
                              hipMemcpyDeviceToHost ) );

    // verify that the GPU did the work we requested
    bool success = true;
    for (int i=0; i<N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
            success = false;
        }
    }
    if (success)    printf( "We did it!\n" );

    // free the memory we allocated on the GPU
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );

    // free the memory we allocated on the CPU
    free( a );
    free( b );
    free( c );

    return 0;
}

