#include "hip/hip_runtime.h"
#include "book.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#define N   1024*1024

__global__ void global_max( int *d_values, int *d_global_max ) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int val = d_values[tid];
	atomicMax(d_global_max, val);
}

int main( void ) {
    int *values;
	int globalMax;
	int golden_globalMax = 0;
    int *dev_values, *dev_globalMax;
    int i;
 
 // allocate the memory on the CPU
    values = (int*)malloc( N * sizeof(int) );
 
    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_values, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_globalMax, sizeof(int) ) );


    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        values[i] = rand();
		if(values[i] > golden_globalMax)
			golden_globalMax = values[i];
    }
    
	
    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_values, values, N * sizeof(int),
                              hipMemcpyHostToDevice ) );

    // Get start time event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
	
    global_max<<<N/256,256>>>( dev_values, dev_globalMax );
	 
	// Get stop time event    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 
    // Compute execution time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
	
	//check cuda error
    hipError_t status = hipGetLastError();
    if ( hipSuccess != status ){
        fprintf(stderr, "Error: %s\n", hipGetErrorString(status));
        exit(1) ;
    }
	
    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( &globalMax, dev_globalMax, sizeof(int),
                              hipMemcpyDeviceToHost ) );

    // verify that the GPU did the work we requested
    bool success = true;
    if (globalMax != golden_globalMax) {
        printf( "globalMax:%d, golden_globalMax: %d\n", globalMax, golden_globalMax);
        success = false;
    }
 
    if (success)    printf( "We did it!\n" );

    // free the memory we allocated on the GPU
    HANDLE_ERROR( hipFree( dev_values ) );
    HANDLE_ERROR( hipFree( dev_globalMax ) );
 

    // free the memory we allocated on the CPU
    free( values );


    return 0;
}

