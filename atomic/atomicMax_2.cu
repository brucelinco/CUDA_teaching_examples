#include "hip/hip_runtime.h"
#include "book.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#define N   1024*1024

__global__ void global_max( int *values, int *global_max, int *local_max, int num_locals ) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int val = values[tid];
	int li = tid % num_locals;
	int old_max = atomicMax(&local_max[li], val);
	if(old_max < val){
		atomicMax(global_max, val);
	}
}

int main( void ) {
    int *values;
	int globalMax;
	int golden_globalMax = 0;
    int *dev_values, *dev_globalMax, *dev_localMax;
    int i;
	int num_locals = 256;
 
 // allocate the memory on the CPU
    values = (int*)malloc( N * sizeof(int) );
 
    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_values, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_globalMax, sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_localMax, num_locals * sizeof(int) ) );

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        values[i] = rand();
		if(values[i] > golden_globalMax)
			golden_globalMax = values[i];
    }
    
	
    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_values, values, N * sizeof(int),
                              hipMemcpyHostToDevice ) );

    // Get start time event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
	
    global_max<<<N/num_locals, num_locals>>>( dev_values, dev_globalMax, dev_localMax, num_locals  );
	 
	// Get stop time event    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 
    // Compute execution time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
	
	//check cuda error
    hipError_t status = hipGetLastError();
    if ( hipSuccess != status ){
        fprintf(stderr, "Error: %s\n", hipGetErrorString(status));
        exit(1) ;
    }
	
    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( &globalMax, dev_globalMax, sizeof(int),
                              hipMemcpyDeviceToHost ) );

    // verify that the GPU did the work we requested
    bool success = true;
    if (globalMax != golden_globalMax) {
        printf( "globalMax:%d, golden_globalMax: %d\n", globalMax, golden_globalMax);
        success = false;
    }
 
    if (success)    printf( "We did it!\n" );

    // free the memory we allocated on the GPU
    HANDLE_ERROR( hipFree( dev_values ) );
    HANDLE_ERROR( hipFree( dev_globalMax ) );
 

    // free the memory we allocated on the CPU
    free( values );


    return 0;
}

